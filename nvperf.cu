#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


struct GpuTimer
{
  hipEvent_t start_;
  hipEvent_t stop_;

  GpuTimer()
  {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
  }

  ~GpuTimer()
  {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  void Start()
  {
    hipEventRecord(start_, 0);
  }

  void Stop()
  {
    hipEventRecord(stop_, 0);
  }

  float Elapsed()
  {
    float elapsed;
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&elapsed, start_, stop_);
    return elapsed;
  }
};

bool float32_perf(bool use_tensorcode)
{
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_T;
    int m = 16384;
    int n = 16384;
    int k = 16384;
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasStatus_t stat = HIPBLAS_STATUS_SUCCESS;
    hipblasHandle_t handle;
    hipblasCreate (&handle);

    if(use_tensorcode==true)
    {
        stat = hipblasSetMathMode (handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    }

    int lda = (transa == HIPBLAS_OP_N) ? max (1, m) : max (1, k);
    int ldb = (transb == HIPBLAS_OP_N) ? max (1, k) : max (1, n);
    int ldc = max (1, m);
    int ka = (transa == HIPBLAS_OP_N) ? k : m;
    int kb = (transb == HIPBLAS_OP_N) ? n : k;
    
    size_t Asz = (size_t)lda * ka * sizeof (float);
    size_t Bsz = (size_t)ldb * kb * sizeof (float);
    size_t Csz = (size_t)ldc * n  * sizeof (float);
    float *A_d = 0, *B_d = 0, *C_d = 0;
    hipMalloc ((void**)&A_d, Asz);
    hipMalloc ((void**)&B_d, Bsz);
    hipMalloc ((void**)&C_d, Csz);
    
    float *A = 0, *B = 0, *C = 0;
    A = (float*) malloc (Asz);
    B = (float*) malloc (Bsz);
    C = (float*) malloc (Csz);
    for (int i = 0; i < lda * ka; i++) A [i] = 1.0f;
    for (int i = 0; i < ldb * kb; i++) B [i] = 2.0f;
    hipMemcpy (A_d, A, Asz, hipMemcpyHostToDevice);
    hipMemcpy (B_d, B, Bsz, hipMemcpyHostToDevice);
    hipMemset (C_d, 0xff, Csz);
    
    hipDeviceSynchronize();
    GpuTimer timer;

    timer.Start();    
    stat = hipblasSgemm(handle, transa, transb, m, n, k, &alpha, A_d, lda, B_d, ldb, &beta, C_d, ldc);
    hipDeviceSynchronize();
    timer.Stop();

    hipMemcpy (C, C_d, Csz, hipMemcpyDeviceToHost);

    auto elapsed = timer.Elapsed();
    auto tflop = 2.0e-9 * m * n *k;

    printf("Implemented CUDA code ran in: %f msecs.\n", timer.Elapsed());
    printf("Performance: %f TFLOPS\n", tflop / elapsed);

    return EXIT_SUCCESS;
}

bool float16_perf()
{
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_T;
    int m = 16384;
    int n = 16384;
    int k = 16384;
    __half alpha = 1.0f;
    __half beta = 0.0f;

    hipblasStatus_t stat = HIPBLAS_STATUS_SUCCESS;
    hipblasHandle_t handle;
    hipblasCreate (&handle);

    stat = hipblasSetMathMode (handle, HIPBLAS_TENSOR_OP_MATH);

    int lda = (transa == HIPBLAS_OP_N) ? max (1, m) : max (1, k);
    int ldb = (transb == HIPBLAS_OP_N) ? max (1, k) : max (1, n);
    int ldc = max (1, m);
    int ka = (transa == HIPBLAS_OP_N) ? k : m;
    int kb = (transb == HIPBLAS_OP_N) ? n : k;
    
    size_t Asz = (size_t)lda * ka * sizeof (__half);
    size_t Bsz = (size_t)ldb * kb * sizeof (__half);
    size_t Csz = (size_t)ldc * n  * sizeof (__half);
    __half *A_d = 0, *B_d = 0, *C_d = 0;
    hipMalloc ((void**)&A_d, Asz);
    hipMalloc ((void**)&B_d, Bsz);
    hipMalloc ((void**)&C_d, Csz);
    
    __half *A = 0, *B = 0, *C = 0;
    A = (__half*) malloc (Asz);
    B = (__half*) malloc (Bsz);
    C = (__half*) malloc (Csz);
    for (int i = 0; i < lda * ka; i++) A [i] = 1.0f;
    for (int i = 0; i < ldb * kb; i++) B [i] = 2.0f;
    hipMemcpy (A_d, A, Asz, hipMemcpyHostToDevice);
    hipMemcpy (B_d, B, Bsz, hipMemcpyHostToDevice);
    hipMemset (C_d, 0xff, Csz);
    
    hipDeviceSynchronize();
    GpuTimer timer;

    timer.Start();    
    stat = hipblasHgemm(handle, transa, transb, m, n, k, &alpha, A_d, lda, B_d, ldb, &beta, C_d, ldc);
    hipDeviceSynchronize();
    timer.Stop();

    hipMemcpy (C, C_d, Csz, hipMemcpyDeviceToHost);

    auto elapsed = timer.Elapsed();
    auto tflop = 2.0e-9 * m * n *k;

    printf("Implemented CUDA code ran in: %f msecs.\n", elapsed);
    printf("Performance: %f TFLOPS\n", tflop / elapsed);

    return EXIT_SUCCESS;
}

bool int8_perf()
{
    using mt = char;
    using rt = int;
    using st = int;
    hipDataType   Atype = HIP_R_8I;
    hipDataType   Ctype = HIP_R_32I;
    hipblasComputeType_t   computeType = HIPBLAS_COMPUTE_32I;

    int dim = 16384;
    int m = dim;
    int n = dim;
    int k = dim;
    mt *A, *B;
    rt *C;
    hipMalloc(&A, sizeof(A[0])*m*k);
    hipMalloc(&B, sizeof(B[0])*n*k);
    hipMalloc(&C, sizeof(C[0])*m*n);
    st alpha = 1;
    st beta = 0;
    hipblasHandle_t h;
    hipblasStatus_t stat = hipblasCreate(&h);

    GpuTimer timer;
    timer.Start();

    stat = hipblasGemmEx(h, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, Atype, dim, 
                        B, Atype, dim, &beta, C, Ctype, dim, computeType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    
    hipDeviceSynchronize();
    timer.Stop();

    auto elapsed = timer.Elapsed();
    auto tflop = 2.0e-9 * m * n *k;

    printf("Implemented CUDA code ran in: %f msecs.\n", elapsed);
    printf("Performance: %f TFLOPS\n", tflop / elapsed);
  // hipError_t err = hipGetLastError();
  // std::cout << hipGetErrorString(err) << std::endl;

  return 0;
}


extern "C" bool runPerf(int device_id)
{   
    bool status = true;
    
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    int driverVersion = 0, runtimeVersion = 0;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
           deviceProp.major, deviceProp.minor);    
    
    printf(
    "  GPU Max Clock rate:                            %.0f MHz\n",
    deviceProp.clockRate * 1e-3f);

#if CUDART_VERSION >= 5000
    // This is supported in CUDA 5.0 (runtime API device properties)
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);
#else
    // This only available in CUDA 4.0-4.2 (but these were only exposed in the
    // CUDA Driver API)
    int memoryClock;
    getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate,
                          dev);
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           memoryClock * 1e-3f);
#endif


    status = float32_perf(false);
    status = float32_perf(true);
    status = float16_perf();
    status = int8_perf();

    return status;
}