#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblaslt.h>
#include <hip/hip_fp8.h>


struct GpuTimer
{
  hipEvent_t start_;
  hipEvent_t stop_;

  GpuTimer()
  {
    hipEventCreate(&start_);
    hipEventCreate(&stop_);
  }

  ~GpuTimer()
  {
    hipEventDestroy(start_);
    hipEventDestroy(stop_);
  }

  void Start()
  {
    hipEventRecord(start_, 0);
  }

  void Stop()
  {
    hipEventRecord(stop_, 0);
  }

  float Elapsed()
  {
    float elapsed;
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&elapsed, start_, stop_);
    return elapsed;
  }
};

bool float32_perf(bool use_tensorcode)
{
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_T;
    int m = 16384;
    int n = 16384;
    int k = 16384;
    float alpha = 1.0f;
    float beta = 0.0f;

    hipblasStatus_t stat = HIPBLAS_STATUS_SUCCESS;
    hipblasHandle_t handle;
    hipblasCreate (&handle);

    if(use_tensorcode==true)
    {
        stat = hipblasSetMathMode (handle, HIPBLAS_TF32_TENSOR_OP_MATH);
    }

    int lda = (transa == HIPBLAS_OP_N) ? max (1, m) : max (1, k);
    int ldb = (transb == HIPBLAS_OP_N) ? max (1, k) : max (1, n);
    int ldc = max (1, m);
    int ka = (transa == HIPBLAS_OP_N) ? k : m;
    int kb = (transb == HIPBLAS_OP_N) ? n : k;
    
    size_t Asz = (size_t)lda * ka * sizeof (float);
    size_t Bsz = (size_t)ldb * kb * sizeof (float);
    size_t Csz = (size_t)ldc * n  * sizeof (float);
    float *A_d = 0, *B_d = 0, *C_d = 0;
    hipMalloc ((void**)&A_d, Asz);
    hipMalloc ((void**)&B_d, Bsz);
    hipMalloc ((void**)&C_d, Csz);
    
    float *A = 0, *B = 0, *C = 0;
    A = (float*) malloc (Asz);
    B = (float*) malloc (Bsz);
    C = (float*) malloc (Csz);
    for (int i = 0; i < lda * ka; i++) A [i] = 1.0f;
    for (int i = 0; i < ldb * kb; i++) B [i] = 2.0f;
    hipMemcpy (A_d, A, Asz, hipMemcpyHostToDevice);
    hipMemcpy (B_d, B, Bsz, hipMemcpyHostToDevice);
    hipMemset (C_d, 0xff, Csz);
    
    hipDeviceSynchronize();
    GpuTimer timer;

    timer.Start();    
    stat = hipblasSgemm(handle, transa, transb, m, n, k, &alpha, A_d, lda, B_d, ldb, &beta, C_d, ldc);
    hipDeviceSynchronize();
    timer.Stop();

    hipMemcpy (C, C_d, Csz, hipMemcpyDeviceToHost);

    auto elapsed = timer.Elapsed();
    auto tflop = 2.0e-9 * m * n *k;

    printf("Implemented CUDA code ran in: %f msecs.\n", timer.Elapsed());
    printf("Performance: %f TFLOPS\n", tflop / elapsed);

    return EXIT_SUCCESS;
}

bool float16_perf()
{
    hipblasOperation_t transa = HIPBLAS_OP_N;
    hipblasOperation_t transb = HIPBLAS_OP_T;
    int m = 16384;
    int n = 16384;
    int k = 16384;
    __half alpha = 1.0f;
    __half beta = 0.0f;

    hipblasStatus_t stat = HIPBLAS_STATUS_SUCCESS;
    hipblasHandle_t handle;
    hipblasCreate (&handle);

    stat = hipblasSetMathMode (handle, HIPBLAS_TENSOR_OP_MATH);

    int lda = (transa == HIPBLAS_OP_N) ? max (1, m) : max (1, k);
    int ldb = (transb == HIPBLAS_OP_N) ? max (1, k) : max (1, n);
    int ldc = max (1, m);
    int ka = (transa == HIPBLAS_OP_N) ? k : m;
    int kb = (transb == HIPBLAS_OP_N) ? n : k;
    
    size_t Asz = (size_t)lda * ka * sizeof (__half);
    size_t Bsz = (size_t)ldb * kb * sizeof (__half);
    size_t Csz = (size_t)ldc * n  * sizeof (__half);
    __half *A_d = 0, *B_d = 0, *C_d = 0;
    hipMalloc ((void**)&A_d, Asz);
    hipMalloc ((void**)&B_d, Bsz);
    hipMalloc ((void**)&C_d, Csz);
    
    __half *A = 0, *B = 0, *C = 0;
    A = (__half*) malloc (Asz);
    B = (__half*) malloc (Bsz);
    C = (__half*) malloc (Csz);
    for (int i = 0; i < lda * ka; i++) A [i] = 1.0f;
    for (int i = 0; i < ldb * kb; i++) B [i] = 2.0f;
    hipMemcpy (A_d, A, Asz, hipMemcpyHostToDevice);
    hipMemcpy (B_d, B, Bsz, hipMemcpyHostToDevice);
    hipMemset (C_d, 0xff, Csz);
    
    hipDeviceSynchronize();
    GpuTimer timer;

    timer.Start();    
    stat = hipblasHgemm(handle, transa, transb, m, n, k, &alpha, A_d, lda, B_d, ldb, &beta, C_d, ldc);
    hipDeviceSynchronize();
    timer.Stop();

    hipMemcpy (C, C_d, Csz, hipMemcpyDeviceToHost);

    auto elapsed = timer.Elapsed();
    auto tflop = 2.0e-9 * m * n *k;

    printf("Implemented CUDA code ran in: %f msecs.\n", elapsed);
    printf("Performance: %f TFLOPS\n", tflop / elapsed);

    return EXIT_SUCCESS;
}

bool int8_perf()
{
    using mt = char;
    using rt = int;
    using st = int;
    hipDataType   Atype = HIP_R_8I;
    hipDataType   Ctype = HIP_R_32I;
    hipblasComputeType_t   computeType = HIPBLAS_COMPUTE_32I;

    int dim = 16384;
    int m = dim;
    int n = dim;
    int k = dim;
    mt *A, *B;
    rt *C;
    hipMalloc(&A, sizeof(A[0])*m*k);
    hipMalloc(&B, sizeof(B[0])*n*k);
    hipMalloc(&C, sizeof(C[0])*m*n);
    st alpha = 1;
    st beta = 0;
    hipblasHandle_t h;
    hipblasStatus_t stat = hipblasCreate(&h);

    GpuTimer timer;
    timer.Start();

    stat = hipblasGemmEx(h, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &alpha, A, Atype, dim, 
                        B, Atype, dim, &beta, C, Ctype, dim, computeType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    
    hipDeviceSynchronize();
    timer.Stop();

    auto elapsed = timer.Elapsed();
    auto tflop = 2.0e-9 * m * n *k;

    printf("Implemented CUDA code ran in: %f msecs.\n", elapsed);
    printf("Performance: %f TFLOPS\n", tflop / elapsed);
  // hipError_t err = hipGetLastError();
  // std::cout << hipGetErrorString(err) << std::endl;

  return 0;
}

bool fp8_perf()
{   
    float alpha = 2.0, beta = 0.0;
    float AscaleHost=2.0, BscaleHost=0.5, CscaleHost=1.0, DscaleHost=1.0, DamaxHost;
    float *a_scale, *b_scale, *c_scale, *d_scale, *amax_d;
    hipblasOperation_t transa = HIPBLAS_OP_T;
    hipblasOperation_t transb = HIPBLAS_OP_N;
    hipblasStatus_t stat = HIPBLAS_STATUS_SUCCESS;
    __hip_fp8_e4m3_fnuz *A, *B, *D;
    int dim = 16384;
    int m = dim;
    int n = dim;
    int k = dim;    
    int lda = (transa == HIPBLAS_OP_N) ? max (1, m) : max (1, k);
    int ldb = (transb == HIPBLAS_OP_N) ? max (1, k) : max (1, n);
    int ldc = max (1, m);    
    hipblasLtMatmulDesc_t operationDesc = NULL;
    hipblasLtMatrixLayout_t Adesc = NULL, Bdesc = NULL, Cdesc = NULL, Ddesc = NULL;
    hipblasLtMatmulPreference_t preference = NULL;
    void *workspace;
    size_t workspaceSize = 12ULL * 1024 * 1024 * 1024;
    hipblasLtHandle_t ltHandle;
    int returnedResults                             = 0;
    hipblasLtMatmulHeuristicResult_t heuristicResult = {};

    hipblasLtCreate(&ltHandle);
    hipMalloc(reinterpret_cast<void**>(&A), m * k * sizeof(__hip_fp8_e4m3_fnuz));
    hipMalloc(reinterpret_cast<void**>(&B), n * k * sizeof(__hip_fp8_e4m3_fnuz));
    hipMalloc(reinterpret_cast<void**>(&D), m * n * sizeof(__hip_fp8_e4m3_fnuz));
    hipMalloc(&workspace, workspaceSize);
    hipMalloc(reinterpret_cast<void**>(&a_scale), sizeof(*a_scale));
    hipMalloc(reinterpret_cast<void**>(&b_scale), sizeof(*b_scale));
    hipMalloc(reinterpret_cast<void**>(&c_scale), sizeof(*c_scale));
    hipMalloc(reinterpret_cast<void**>(&d_scale), sizeof(*d_scale));
    hipMalloc(reinterpret_cast<void**>(&amax_d), sizeof(*amax_d));

    std::vector<__hip_fp8_e4m3_fnuz> Ahost(m*k), Bhost(n*k);
    std::vector<__hip_fp8_e4m3_fnuz> Chost(m*n), biasHost(m);
    for (int i = 0; i < m * k; i++) Ahost[i] = __hip_fp8_e4m3_fnuz(i);
    for (int i = 0; i < n * k; i++) Bhost[i] = __hip_fp8_e4m3_fnuz(i);
    for (int i = 0; i < m; i++) biasHost[i] = __hip_fp8_e4m3_fnuz(i + 1);

    hipMemcpyAsync(A, Ahost.data(), Ahost.size() * sizeof(Ahost[0]), hipMemcpyHostToDevice);
    hipMemcpyAsync(B, Bhost.data(), Bhost.size() * sizeof(Bhost[0]), hipMemcpyHostToDevice);
    hipMemcpyAsync(a_scale, &AscaleHost, sizeof(AscaleHost), hipMemcpyHostToDevice);
    hipMemcpyAsync(b_scale, &BscaleHost, sizeof(BscaleHost), hipMemcpyHostToDevice);
    hipMemcpyAsync(c_scale, &CscaleHost, sizeof(CscaleHost), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_scale, &DscaleHost, sizeof(DscaleHost), hipMemcpyHostToDevice);
    hipMemcpyAsync(amax_d, &DamaxHost, sizeof(DamaxHost), hipMemcpyHostToDevice);

    // create operation desciriptor; see hipblasLtMatmulDescAttributes_t for details about defaults; here we just need to
    // set the transforms for A and B
    hipblasLtMatmulDescCreate(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));

    // set scaling factors
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER, &a_scale, sizeof(a_scale));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER, &b_scale, sizeof(b_scale));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER, &c_scale, sizeof(c_scale));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER, &d_scale, sizeof(d_scale));
    hipblasLtMatmulDescSetAttribute(operationDesc, HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER, &amax_d, sizeof(amax_d));

    // create matrix descriptors, we are good with the details here so no need to set any extra attributes
    // table of supported type combinations can be found in the documentation: https://docs.nvidia.com/cuda/cublas/index.html#cublasltmatmul
    hipblasLtMatrixLayoutCreate(&Adesc, HIP_R_8F_E4M3, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
    hipblasLtMatrixLayoutCreate(&Bdesc, HIP_R_8F_E4M3, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
    hipblasLtMatrixLayoutCreate(&Cdesc, HIP_R_16BF, m, n, ldc);
    hipblasLtMatrixLayoutCreate(&Ddesc, HIP_R_8F_E4M3, m, n, ldc);

    // create preference handle; here we could use extra attributes to disable tensor ops or to make sure algo selected
    // will work with badly aligned A, B, C; here for simplicity we just assume A,B,C are always well aligned (e.g.
    // directly come from hipMalloc)
    hipblasLtMatmulPreferenceCreate(&preference);
    hipblasLtMatmulPreferenceSetAttribute(preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));

    // we just need the best available heuristic to try and run matmul. There is no guarantee this will work, e.g. if A
    // is badly aligned, you can request more (e.g. 32) algos and try to run them one by one until something works
    hipblasLtMatmulAlgoGetHeuristic(ltHandle, operationDesc, Adesc, Bdesc, Cdesc, Ddesc, preference, 1, &heuristicResult, &returnedResults);

    GpuTimer timer;
    timer.Start();

    stat = hipblasLtMatmul(ltHandle, operationDesc, &alpha, A, Adesc, B, Bdesc, &beta,
                          nullptr, Cdesc, D, Ddesc, &heuristicResult.algo,
                          workspace, workspaceSize, 0);
    
    hipDeviceSynchronize();
    timer.Stop();

    auto elapsed = timer.Elapsed();
    auto tflop = 2.0e-9 * m * n *k;

    printf("Implemented CUDA code ran in: %f msecs.\n", elapsed);
    printf("Performance: %f TFLOPS\n", tflop / elapsed);
    // hipError_t err = hipGetLastError();
    // std::cout << hipGetErrorString(err) << std::endl;

  return 0;
}


extern "C" bool runPerf(int device_id)
{   
    bool status = true;
    
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    int driverVersion = 0, runtimeVersion = 0;
    hipDriverGetVersion(&driverVersion);
    hipRuntimeGetVersion(&runtimeVersion);
    printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
           driverVersion / 1000, (driverVersion % 100) / 10,
           runtimeVersion / 1000, (runtimeVersion % 100) / 10);
    printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
           deviceProp.major, deviceProp.minor);    
    
    printf(
    "  GPU Max Clock rate:                            %.0f MHz\n",
    deviceProp.clockRate * 1e-3f);

#if CUDART_VERSION >= 5000
    // This is supported in CUDA 5.0 (runtime API device properties)
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           deviceProp.memoryClockRate * 1e-3f);
#else
    // This only available in CUDA 4.0-4.2 (but these were only exposed in the
    // CUDA Driver API)
    int memoryClock;
    getCudaAttribute<int>(&memoryClock, hipDeviceAttributeMemoryClockRate,
                          dev);
    printf("  Memory Clock rate:                             %.0f Mhz\n",
           memoryClock * 1e-3f);
#endif


    status = float32_perf(false);
    status = float32_perf(true);
    status = float16_perf();
    status = int8_perf();
    status = fp8_perf();

    return status;
}
